#include "include/utils.h"

extern "C" {
using namespace simple_cuda;

/**
 * @brief Measures the execution time of a CUDA kernel.
 *
 * This function measures the execution time of a CUDA kernel by recording
 * CUDA events before and after the kernel launch. The time is measured in
 * milliseconds.
 *
 * @param kernelLauncher A std::function object that launches the CUDA kernel.
 * This function should take no arguments and have no return value. The
 * function should launch the CUDA kernel with the desired grid and block
 * dimensions.
 *
 * @return The execution time of the CUDA kernel in milliseconds.
 *
 * @note The caller is responsible for ensuring that any device memory
 * accessed by the kernel is allocated before this function is called and
 * deallocated after this function returns. The caller is also responsible
 * for handling any CUDA errors that may occur during the kernel execution.
 */
extern "C" float kernel_time(std::function<void()> kernelLauncher) {
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, 0);
  kernelLauncher();

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);

  hipEventDestroy(start);
  hipEventDestroy(stop);
  return elapsedTime;
}

} // extern "C"