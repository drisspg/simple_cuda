#include "include/utils.h"

using namespace simple_cuda;

float kernel_time(std::function<void()> kernelLauncher) {
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, 0);
  kernelLauncher();

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);

  hipEventDestroy(start);
  hipEventDestroy(stop);
  return elapsedTime;
}