#include "hip/hip_runtime.h"
#include "src/include/tensors.h"
#include "src/include/utils.h"

#include <cstddef>
#include <fmt/core.h>
#include <fmt/ranges.h>

#include <ranges>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/host_vector.h>

using namespace simple_cuda;
using KernelFunc = void (*)(const float *, const float *, float *, int);

using two_d = Extent<2>;

template <int tile_size, int filter_radius>
__global__ void Conv2D(const float *input, const float *filter, float *output,
                       const int numel) {
  const int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (global_idx >= numel) {
    // Early return for invalid output elements
    return;
  }

  float accumulator = 0.0;
#pragma unroll
  for (int j{-filter_radius}; j <= filter_radius; j++) {
    if (0 < global_idx + j < numel) {
      accumulator += input[global_idx + j] * filter[j + filter_radius];
    }
  }
  // We early returned so we can write freely
  output[global_idx] += accumulator;
}

template <int tile_size, int filter_radius>
__device__ void fill_tile(const float *input, float *input_tile,
                          const int global_idx, const int input_size) {
  // Load the non ghost cells
  input_tile[threadIdx.x + filter_radius] =
      global_idx < input_size ? input[global_idx] : 0.0;

  // Edges load in ghost cells
  if (static_cast<int>(threadIdx.x - filter_radius) < 0) {
    const int offset = global_idx - filter_radius;
    input_tile[threadIdx.x] = offset >= 0 ? input[offset] : 0.0;
  }
  if (threadIdx.x + filter_radius >= tile_size) {
    const int offset = global_idx + filter_radius;
    input_tile[filter_radius + threadIdx.x + filter_radius] =
        offset < input_size ? input[offset] : 0.0;
  }
}

template <int tile_size, int filter_radius>
__global__ void Conv1D_shmem(const float *input, const float *filter,
                             float *output, const int input_size) {
  const int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
  // We stored the the tile_size which is equivelant to block size + the filter
  // size for the left and right boundaries
  __shared__ float input_tile[tile_size + (2 * filter_radius)];

  fill_tile<tile_size, filter_radius>(input, input_tile, global_idx,
                                      input_size);
  __syncthreads();
  float accumulator = 0.0;
#pragma unroll
  for (int j{-filter_radius}; j <= filter_radius; j++) {
    const int tile_idx = filter_radius + threadIdx.x + j;
    accumulator += input_tile[tile_idx] * filter[j + filter_radius];
  }

  if (global_idx < input_size) {
    output[global_idx] = accumulator;
  }
}

template <typename T>
T cpp_kernel(T const &input, T const &filter, const int n_rows,
             const int n_cols, const int filter_radius) {
  std::vector<float> output;
  output.reserve(input.size());
  for (int i{0}; i < n_rows; i++) {
    for (int j{0}; j < n_cols; j++) {
      float accum = 0.0;
      for (int k{-filter_radius}; k <= filter_radius; k++) {
        for (int l{-filter_radius}; l <= filter_radius; l++) {
          // if (0 <= i + j && i + j < n_rows && 0 <= j + l && j + l < n_cols) {
            // accum += input[i + j] * filter[j + kernel_width];
          // }
        }
      }
      output.emplace_back(accum);
    }
  }

  return output;
}

void Test(KernelFunc func, const size_t n_rows, const size_t n_cols,
          const size_t filter_radius, dim3 grid, dim3 block) {
  two_d tensor_extents({n_rows, n_cols});
  two_d filter_extents({(2 * filter_radius) * (2 * filter_radius)});

  HostTensor<float, two_d> input_vec(tensor_extents);
  HostTensor<float, two_d> output_vec(tensor_extents);
  HostTensor<float, two_d> filter(filter_extents);

  std::fill(input_vec.data_.begin(), input_vec.data_.end(), 1);
  std::fill(output_vec.data_.begin(), output_vec.data_.end(), 0);
  std::fill(filter.data_.begin(), filter.data_.end(), 2);

  auto input_vec_d = input_vec.to_device();
  auto output_vec_d = output_vec.to_device();
  auto filter_d = filter.to_device();

  func<<<grid, block>>>(input_vec_d.data_ptr(), filter_d.data_ptr(),
                        output_vec_d.data_ptr(), tensor_extents.numel());
  cudaCheckErrors("kernel launch failure");
  hipDeviceSynchronize();

  auto host_output = output_vec_d.to_host();
  auto host_output_ptr = host_output.data_ptr();

  const auto cpp_anwser =
      cpp_kernel(input_vec.data_, filter.data_, n_rows, n_cols, filter_radius);

  for (const int idx : std::views::iota(0, int(tensor_extents.numel()))) {
    if (host_output_ptr[idx] != cpp_anwser[idx]) {
      std::string error_string = "Houston we have a problem!\n";
      error_string += fmt::format("At ({}) found value: {} instead of {}!\n",
                                  idx, host_output_ptr[idx], cpp_anwser[idx]);
      std::cout << error_string;

      if (tensor_extents.numel() <= 32) {
        fmt::print("Good:{}\n", fmt::join(cpp_anwser, ", "));
        fmt::print("Bad:{}\n", fmt::join(host_output.data_, ", "));
      }

      exit(1);
    }
  }
  std::cout << "All good brother!\n";
}

int main() {
  // Standard Matmul
  constexpr int num_rows = 256;
  constexpr int num_cols = 256;
  constexpr int filter_radius = 3;
  constexpr int block_size = 32;

  // dimx is inner dim, dimy is outerdim
  dim3 grid(ceil_div(num_rows, block_size), ceil_div(num_cols, block_size));
  dim3 block(block_size, block_size);

  Test(Conv2D<block_size, filter_radius>, num_rows, num_cols, filter_radius,
       grid, block);

  // Test(Conv1D_shmem<block_size, filter_radius>, max_length, max_length,
  // filter_radius, grid,
  //      block);

  // profile the relevant kernels:
  // ncu -k "regex:Conv" ./bin/conv1d
  return 0;
}