
#include <hip/hip_runtime.h>
#include <hip/hip_fp8.h>
#include <stdio.h>

__global__ void convert_to_e8m0(float *in, __hip_fp8_storage_t *out) {
  const float input_val = in[0];
  printf("Device input value: %f\n", input_val);
  __hip_fp8_storage_t result =
      __nv_cvt_float_to_e8m0(input_val, __HIP_SATFINITE, hipRoundNearest);
  printf("Device output value (hex): 0x%02x, (decimal): %u\n",
         (unsigned char)result, (unsigned char)result);
  out[0] = result;
}

int main() {
  float h_in = 1.0f / 448.0f;
  float *d_in;
  __hip_fp8_storage_t *d_out, h_out;

  hipMalloc(&d_in, sizeof(float));
  hipMalloc(&d_out, sizeof(__hip_fp8_storage_t));

  hipError_t err =
      hipMemcpy(d_in, &h_in, sizeof(float), hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    printf("Memcpy error: %s\n", hipGetErrorString(err));
    return 1;
  }

  convert_to_e8m0<<<1, 1>>>(d_in, d_out);
  hipDeviceSynchronize(); // Need this to see printf from kernel
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("Kernel error: %s\n", hipGetErrorString(err));
    return 1;
  }

  err = hipMemcpy(&h_out, d_out, sizeof(__hip_fp8_storage_t),
                   hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    printf("Memcpy error: %s\n", hipGetErrorString(err));
    return 1;
  }

  printf("Host input float: %f\n", h_in);
  printf("Host output e8m0 hex: 0x%02x, decimal: %u\n", (unsigned char)h_out,
         (unsigned char)h_out);
  printf("Host output e8m0 bits: ");
  for (int i = 7; i >= 0; i--) {
    printf("%d", (h_out >> i) & 0x1);
  }
  printf("\n");

  hipFree(d_in);
  hipFree(d_out);
  return 0;
}
