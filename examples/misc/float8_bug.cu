#include "hip/hip_runtime.h"
#include <cstdint>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>
#include <functional>
#include <stdio.h>
#include <type_traits>
#include <fmt/core.h>
#include <bitset>
#include <iostream>

// error checking macro
#define cudaCheckErrors(msg)                                                   \
  do {                                                                         \
    hipError_t __err = hipGetLastError();                                    \
    if (__err != hipSuccess) {                                                \
      fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", msg,                  \
              hipGetErrorString(__err), __FILE__, __LINE__);                  \
      fprintf(stderr, "*** FAILED - ABORTING\n");                              \
      exit(1);                                                                 \
    }                                                                          \
  } while (0)


__global__ void saturated_cast_kernel_single(
   const float *input, __hip_fp8_storage_t *output, int n_rows, int n_cols,
    __hip_fp8_interpretation_t out_dtype, float *scaler) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  // Assume row major
  const int global_index = row * n_cols + col;
  if (row < n_rows && col < n_cols) {
      const float scaled_input = input[global_index] * (*scaler);
      output[global_index] = __hip_cvt_float_to_fp8(
          scaled_input, __hip_saturation_t::__HIP_SATFINITE, out_dtype);
  }
}

int main() {
  float inpt[2] = {0.3223, 0.3223};
  float scale = 57344.0;
  float output[2] = {0.0, 0.0};

  // Pointer to device array
  float *d_inpt = nullptr;
  float *d_scale = nullptr;
  __hip_fp8_storage_t *d_output = nullptr;
  // Allocate memory on the device
  hipMalloc((void **)&d_inpt, 2 * sizeof(float));
  hipMalloc((void **)&d_scale, sizeof(float));
  hipMalloc((void **)&d_output, 2 * sizeof(__hip_fp8_interpretation_t));
  cudaCheckErrors("hipMalloc failure");

  hipMemcpy(d_inpt, inpt, 2 * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_scale, &scale, sizeof(float), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy H2D failure");

  dim3 grid(1,1);
  dim3 block(2, 2);

  saturated_cast_kernel_single<<<grid, block>>>(
      static_cast<float *>(d_inpt), static_cast<__hip_fp8_storage_t *>(d_output),
      1, 2, __hip_fp8_interpretation_t::__HIP_E5M2_FNUZ,
      static_cast<float *>(d_scale));
  hipDeviceSynchronize();
  cudaCheckErrors("kernel launch failure");
  hipMemcpy(output, d_output, 2 * sizeof(__hip_fp8_storage_t),
             hipMemcpyDeviceToHost);
  fmt::print("Output: {} {}\n", static_cast<uint8_t>(output[0]), static_cast<uint8_t>(output[1]));
  return 0;
}