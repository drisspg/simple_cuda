#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <fmt/core.h>

using namespace cooperative_groups;

__global__ void direct_copy_optimized(int4 *output, int4 *input, size_t n) {
    const size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t stride = blockDim.x * gridDim.x;

    for (size_t i = tid; i < n / 4; i += stride) {
        output[i] = input[i];
    }
}

bool check_equal(int *output, int *input, int n) {
  for (int i = 0; i < n; i++) {
    if (output[i] != input[i]) {
      fmt::print("Not equal for {}, input: {} output: {}\n", i, input[i], output[i]);
      return false;
    }
  }
  return true;
}

int main() {

  int n = 1 << 24;
  int blockSize = 1024;
  int numSMs;
  hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);
  //  manual Grid_size
  int nBlocks_manual = 32 * numSMs;
  int *output, *data;
  hipMallocManaged(&output, n * sizeof(int));
  hipMallocManaged(&data, n * sizeof(int));
  std::fill_n(data, n, 1); // initialize data

  direct_copy_optimized<<<nBlocks_manual, blockSize>>>(reinterpret_cast<int4*>(output), reinterpret_cast<int4*>(data), n);
  hipDeviceSynchronize();

  auto eq = check_equal(output, data, n);
  fmt::print("Equal: {}\n", eq);

  return 0;
}
